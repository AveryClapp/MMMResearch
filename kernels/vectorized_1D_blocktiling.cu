#pragma once

/* Kernel #6, Transposing A matrix into SMEM for better memory access */

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void __launch_bounds__((BM * BN) / (TM * TN), 1)
	vectorized_blocktiling(int M, int N, int K, float *A,
					 float *B, float *C) {
	// Determines where the block will start
	const uint cRow = blockIdx.y;
	const uint cCol = blockIdx.x;

	// We calculate BM * BN elements per block, must find how many threads are
	// needed total (including both dimensions)
	const uint totalResultsBlocktile = BM;
	const uint numThreadsBlocktile = totalResultsBlocktile / TM;

	assert(numThreadsBlocktile == blockDim.x);

	// Blocked groups of cols and sequential rows
	// Assign threadCol and threadRow in row-major order 
	const int threadCol = threadIdx.x % BN;
	const int threadRow = threadIdx.x / BN;

	__shared__ float As[BM * BK];
	__shared__ float Bs[BK * BN];

	// Advance matrix pointers to the start of the block
	A += cRow * BM * K;
	B += cCol * BN;
	C += cRow * BM * N + cCol * BN;

	const uint innerRowA = threadIdx.x / (BK / 4);
	const uint innerColA = threadIdx.x % (BK / 4);
  	const uint innerRowB = threadIdx.x / (BN / 4);
	const uint innerColB = threadIdx.x % (BN / 4);

	float threadResults[TM] = {0.0};

	// Every advance the block through the matrix
	for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
		float4 tmp = 
		  reinterpret_cast<const float4 *>(&A[innerRowA * K + innerColA * 4])[0];
		
		// Load elements from row major order in A to column major in As
		As[(innerColA * 4 + 0) * BM + innerRowA] = tmp.x;
   		As[(innerColA * 4 + 1) * BM + innerRowA] = tmp.y;
	    As[(innerColA * 4 + 2) * BM + innerRowA] = tmp.z;
		As[(innerColA * 4 + 3) * BM + innerRowA] = tmp.w;

		// Load the float4 value from global memory
		reinterpret_cast<float4 *>(&Bs[innerRowB * BN + innerColB * 4])[0] =
		  reinterpret_cast<float4 *>(&B[innerRowB * N + innerColB * 4])[0];
		__syncthreads();

		// Advance the matrix pointers to the start of the next block
		A += BK;
		B += BK * N;			

		for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
			float tmpB = Bs[dotIdx * BN + threadCol];
			for (uint resIdx = 0; resIdx < TM; ++resIdx) {
				/* As is now transposed, so rearrange thread indexing */
				threadResults[resIdx] += As[dotIdx * TM + (threadRow * TM + resIdx)] * tmpB;
			}
		}
		__syncthreads();
	}
	
	// Accumulate results from thread results registerfile into C
	for (uint resIdxM = 0; resIdxM < TM; resIdxM += 1) {
		for (uint resIdxN = 0; resIdxN < TN; resIdxN += 4) {
			float4 tmp;
			tmp.x = threadResults[resIdxM * TN + resIdxN];
			tmp.y = threadResults[resIdxM * TN + resIdxN + 1];
			tmp.z = threadResults[resIdxM * TN + resIdxN + 2];
			tmp.w = threadResults[resIdxM * TN + resIdxN + 3];
			reinterpret_cast<float4 *>(&C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN])[0] = tmp;
		}
	}
}
